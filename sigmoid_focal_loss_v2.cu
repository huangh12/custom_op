#include "hip/hip_runtime.h"
#include "./sigmoid_focal_loss_v2-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>
#include "../common/cuda_utils.h"
#include "mxnet_op.h"
#include <assert.h>

namespace mshadow {
namespace cuda {

template<typename Dtype>
__global__ void SigmoidFocalLossV2Kernel(
    const Dtype* bottom_data, const Dtype* bottom_label, const Dtype* bottom_label_weight, Dtype* top_prob_data, Dtype* top_loss_data,
    const int count, const int channel, const int spatial_dim,
    const float ignore_label, const float nonfocal_label, const float alpha, const float gamma) {

  for (int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       i < count; i += blockDim.x * gridDim.x * gridDim.y) {

    int s = i % spatial_dim;
    int c = (i / spatial_dim) % channel; 
    int n = i / (spatial_dim * channel);    
 
    int t = bottom_label[n * spatial_dim + s]; 
    float w = bottom_label_weight[n * spatial_dim + s]; 

    // p = 1. / 1. + expf(-x)
    float p = 1. / (1. + expf(-bottom_data[i]));

    top_prob_data[i] = p;

    if (t == ignore_label) {
      top_loss_data[i] = 0.0;
    } else if (t == nonfocal_label) {
      // assert(t == c + 1);
      // printf("t: %d, c: %d\n", t,c+1);
      top_loss_data[i] = - w * alpha * logf(max(p, FLT_MIN));
      // printf("Foreground(nonfocal) grad: %f\n", (float) top_loss_data[i]);
    } else if (t == c + 1) {     
      // (1 - p)**gamma * log(p) where
      float term1 = powf((1. - p), gamma) * logf(max(p, FLT_MIN));
      top_loss_data[i] = - w * alpha * term1;
      // printf("Foreground grad: %f\n", (float) top_loss_data[i]);
    } else { // t != c + 1
      // p**gamma * log(1 - p)
      float term2 = powf(p, gamma) * (-1. * bottom_data[i] * (bottom_data[i] >= 0) -
           logf(1. + expf(bottom_data[i] - 2. * bottom_data[i] * (bottom_data[i] >= 0))));      
      top_loss_data[i] = - w * (1.0 - alpha) * term2;
      // printf("Background grad: %f\n", (float) top_loss_data[i]);
    }
  }
}


template<typename Dtype>
inline void SigmoidFocalLossV2Forward(const Tensor<gpu, 3, Dtype> &out_prob,
                                    const Tensor<gpu, 3, Dtype> &out_loss,
                                    const Tensor<gpu, 3, Dtype> &in_data,
                                    const Tensor<gpu, 2, Dtype> &in_label,
                                    const Tensor<gpu, 2, Dtype> &in_label_weight,
                                    const float ignore_label,
                                    const float nonfocal_label,
                                    const float alpha,
                                    const float gamma) {
  const Dtype *bottom_data = in_data.dptr_;
  const Dtype *bottom_label = in_label.dptr_;
  const Dtype *bottom_label_weight = in_label_weight.dptr_;
  Dtype *top_prob_data = out_prob.dptr_;
  Dtype *top_loss_data = out_loss.dptr_;

  const int count = out_prob.shape_.Size();
  const int channel = in_data.size(1);
  const int spatial_dim = in_data.size(2);

  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridDim, (gridSize + kMaxGridDim - 1) / kMaxGridDim);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "SigmoidFocalLossV2 Forward");
  hipStream_t stream = Stream<gpu>::GetStream(out_prob.stream_);

  SigmoidFocalLossV2Kernel<Dtype><<<dimGrid, dimBlock, 0, stream>>>(
      bottom_data, bottom_label, bottom_label_weight, top_prob_data, top_loss_data,
      count, channel, spatial_dim, ignore_label, nonfocal_label, alpha, gamma);
}

template<typename Dtype>
__global__ void SigmoidFocalLossV2GradientKernel(
    const Dtype* bottom_data, const Dtype* bottom_label, const Dtype* bottom_label_weight, Dtype* bottom_data_diff,
    const int count, const int channel, const int spatial_dim,
    const float ignore_label, const float nonfocal_label, const float alpha, const float gamma) {

  for (int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       i < count; i += blockDim.x * gridDim.x * gridDim.y) {

    int s = i % spatial_dim;
    int c = (i / spatial_dim) % channel; 
    int n = i / (spatial_dim * channel);    
 
    int t = bottom_label[n * spatial_dim + s]; 
    float w = bottom_label_weight[n * spatial_dim + s]; 

    // p = 1. / 1. + expf(-x)
    float p = 1. / (1. + expf(-bottom_data[i]));

    if (t == ignore_label) {
      bottom_data_diff[i] = 0.0;
    } else if (t == nonfocal_label) {
      assert(t == c + 1);
      bottom_data_diff[i] = - w * alpha * (1. - p);
    } else if (t == c + 1) {     
      // (1-p)**g * (1 - p - g*p*log(p))
      float term1 =
          powf((1. - p), gamma) *
          (1. - p - (p * gamma * logf(max(p, FLT_MIN))));
      bottom_data_diff[i] = - w * alpha * term1;
    } else { // t != c + 1
      // (p**g) * (g*(1-p)*log(1-p) - p)
      float term2 =
          powf(p, gamma) *
          ((-1. * bottom_data[i] * (bottom_data[i] >= 0) -
           logf(1. + expf(bottom_data[i] - 2. * bottom_data[i] * (bottom_data[i] >= 0)))) *
           (1. - p) * gamma - p);     
      bottom_data_diff[i] = - w * (1.0 - alpha) * term2;
    }
  }
}

template<typename Dtype>
inline void SigmoidFocalLossV2Backward(const Tensor<gpu, 3, Dtype> &in_data_grad,
                                     const Tensor<gpu, 3, Dtype> &in_data,
                                     const Tensor<gpu, 2, Dtype> &in_label,
                                     const Tensor<gpu, 2, Dtype> &in_label_weight,
                                     const float ignore_label,
                                     const float nonfocal_label,
                                     const float alpha,
                                     const float gamma) {
  const Dtype *bottom_data = in_data.dptr_;
  const Dtype* bottom_label = in_label.dptr_;
  const Dtype* bottom_label_weight = in_label_weight.dptr_;
  Dtype *bottom_data_diff = in_data_grad.dptr_;
  
  const int count = in_data_grad.shape_.Size();
  const int channel = in_data.size(1);
  const int spatial_dim = in_data.size(2);
  
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridDim, (gridSize + kMaxGridDim - 1) / kMaxGridDim);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "SigmoidFocalLossV2 Backward");
  hipStream_t stream = Stream<gpu>::GetStream(in_data_grad.stream_);

  SigmoidFocalLossV2GradientKernel<Dtype><<<dimGrid, dimBlock, 0, stream>>>(
      bottom_data, bottom_label, bottom_label_weight, bottom_data_diff, count, channel, spatial_dim, 
      ignore_label, nonfocal_label, alpha, gamma);
}

}  // namespace cuda

template<typename Dtype>
inline void SigmoidFocalLossV2Forward(const Tensor<gpu, 3, Dtype> &out_prob,
                                    const Tensor<gpu, 3, Dtype> &out_loss,
                                    const Tensor<gpu, 3, Dtype> &in_data,
                                    const Tensor<gpu, 2, Dtype> &in_label,
                                    const Tensor<gpu, 2, Dtype> &in_label_weight,
                                    const float ignore_label,
                                    const float nonfocal_label,
                                    const float alpha,
                                    const float gamma) {
  cuda::SigmoidFocalLossV2Forward(out_prob, out_loss, in_data, in_label, in_label_weight, ignore_label, nonfocal_label, alpha, gamma);
}

template<typename Dtype>
inline void SigmoidFocalLossV2Backward(const Tensor<gpu, 3, Dtype> &in_data_grad,
                                     const Tensor<gpu, 3, Dtype> &in_data,
                                     const Tensor<gpu, 2, Dtype> &in_label,
                                     const Tensor<gpu, 2, Dtype> &in_label_weight,
                                     const float ignore_label,
                                     const float nonfocal_label,
                                     const float alpha,
                                     const float gamma) {
  cuda::SigmoidFocalLossV2Backward(in_data_grad, in_data, in_label, in_label_weight, ignore_label, nonfocal_label, alpha, gamma);
}

}  // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(SigmoidFocalLossV2Param param, int dtype) {
  Operator* op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new SigmoidFocalLossV2Op<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
